// Program for Parallel Binary Search in CUDA
// For Hadoop-CUDA Lab

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#include <assert.h>

__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {

	// Integer division trick to round up
	return (((set_size + num_threads) / num_threads) * thread) + offset;
}

__global__ void p_ary_search(int search, int array_length,  int *arr, int *ret_val ) {

	const int num_threads = blockDim.x * gridDim.x;
	const int thread = blockIdx.x * blockDim.x + threadIdx.x;
	
	//ret_val[0] = -1;
	//ret_val[1] = offset;

	int set_size = array_length;

	
	while(set_size != 0){
		// Get the offset of the array, initially set to 0
		int offset = ret_val[1];
		
		// I think this is necessary in case a thread gets ahead, and resets offset before it's read
		// This isn't necessary for the unit tests to pass, but I still like it here
		__syncthreads();	

		// Get the next index to check
		int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

		// If the index is outside the bounds of the array then lets not check it
		if (index_to_check < array_length){

			// If the next index is outside the bounds of the array, then set it to maximum array size
			int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

			if (next_index_to_check >= array_length){
				next_index_to_check = array_length - 1;
			}

			// If we're at the mid section of the array reset the offset to this index
			if (search > arr[index_to_check] && (search < arr[next_index_to_check])) {
				ret_val[1] = index_to_check;
			}
			else if (search == arr[index_to_check]) {
				// Set the return var if we hit it
				ret_val[0] = index_to_check;
			}	
		}

		// Since this is a p-ary search divide by our total threads to get the next set size
		set_size = set_size / num_threads;
		
		// Sync up so no threads jump ahead and get a bad offset
		__syncthreads();
	}
}


int chop_position(int search, int *search_array, int array_length)
{
	// Get the size of the array for future use
	int array_size = array_length * sizeof(int);

	// Don't bother with small arrays
	if (array_size == 0) return -1;

	// Setup array to use on device
    int    *dev_arr;
	hipMalloc((void**)&dev_arr, array_size);

	// Copy search array values
	hipMemcpy(dev_arr, search_array, array_size, hipMemcpyHostToDevice);

	// return values here and on device
	int		*ret_val = (int*)malloc(sizeof(int) * 2);
	ret_val[0] = -1; // return value
	ret_val[1] = 0; // offset
	array_length = array_length % 2 == 0 ? array_length : array_length - 1; // array size

	int		*dev_ret_val;
	hipMalloc((void**)&dev_ret_val, sizeof(int) * 2);

	// Send in some intialized values
	hipMemcpy(dev_ret_val, ret_val, sizeof(int) * 2, hipMemcpyHostToDevice);
	
	// Launch kernel
	// This seems to be the best combo for p-ary search
	// Optimized around 10-15 registers per thread
	p_ary_search<<<16, 64>>>(search, array_length, dev_arr, dev_ret_val);

	// Get results
	hipMemcpy(ret_val, dev_ret_val, 2 * sizeof(int), hipMemcpyDeviceToHost);

	int ret = ret_val[0];

	printf("Ret Val %i    Offset %i\n", ret, ret_val[1]);

	// Free memory on device
	hipFree(dev_arr);
	hipFree(dev_ret_val);
	
	free(ret_val);

	return ret;
}

// Test region
static int * build_array(int length) {

	int *ret_val = (int*)malloc(length * sizeof(int));

	for (int i = 0; i < length; i++)
	{
		ret_val[i] = i * 2 - 1;
	}

	return ret_val;
}

static void test_array(int length, int search, int index) {
	
	printf("Length %i   Search %i    Index %i\n", length, search, index);
	// assert(index == chop_position(search, build_array(length), length) && "test_small_array()");

}

static void test_arrays() {	
	
	test_array(200, 200, -1);
	
	test_array(200, -1, 0);
	
	test_array(200, 1, 1);
	
	test_array(200, 29, 15);
	
	test_array(200, 129, 65);	

	test_array(200, 395, 198);
	
	test_array(20000, 395, 198);
	
	test_array(2000000, 394, -1);
	
	test_array(20000000, 394, -1);
}


int main(){
	test_arrays();
}
