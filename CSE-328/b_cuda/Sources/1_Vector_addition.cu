// Program for Parallel Vector Addition in CUDA
// For Hadoop-CUDA Lab

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 1024         // size of array

__global__ void add(int *a,int *b, int *c) {
	int tid = blockIdx.x *  blockDim.x + threadIdx.x;
        if(tid < N){
          c[tid] = a[tid]+b[tid];
        }
}

int main(int argc, char *argv[])  {
	int T = 10, B = 1;            // threads per block and blocks per grid, taking default values
	int a[N],b[N],c[N];
	int *dev_a, *dev_b, *dev_c;

	printf("Size of array = %d\n", N);
	do {
		printf("Enter number of threads per block: ");
		scanf("%d",&T);
		printf("\nEnter nuumber of blocks per grid: ");
		scanf("%d",&B);
		if (T * B != N) printf("Error T x B != N, try again");
	} while (T * B != N);

	hipEvent_t start, stop;     // using cuda events to measure time
	float elapsed_time_ms;       // which is applicable for asynchronous code also

	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));
	hipMalloc((void**)&dev_c,N * sizeof(int));

	for(int i=0;i<N;i++) {    // load arrays with some numbers
		a[i] = i;
		b[i] = i*1;
	}

	hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);

	hipEventCreate( &start );     // instrument code to measure start time
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	add<<<B,T>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);

	hipEventRecord( stop, 0 );     // instrument code to measue end time
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsed_time_ms, start, stop );

	for(int i=0;i<N;i++) {
		printf("%d+%d=%d\n",a[i],b[i],c[i]);
	}

	printf("Time to calculate results: %f ms.\n", elapsed_time_ms);  // print out execution time

	// clean up
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

