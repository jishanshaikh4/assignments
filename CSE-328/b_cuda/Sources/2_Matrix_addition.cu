// Program for Matrix Addition in CUDA
// For Hadoop-CUDA Lab

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrixadd(int *a,int *b, int *c, int N) {

	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int index = row * N + col;

      	if(col < N && row < N)
          c[index] = a[index]+b[index];

}

void cpu_matrixadd(int *a,int *b, int *c, int N) {

	int index;
	for(int col=0;col < N; col++) 
		for(int row=0;row < N; row++) {
			index = row * N + col;
           		c[index] = a[index]+b[index];
		}
}

int main(int argc, char *argv[])  {

	char key;

	int i, j; 					// loop counters

	int Grid_Dim_x=1, Grid_Dim_y=1;			//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;		//Block structure values

	int noThreads_x, noThreads_y;		// number of threads available in device, each dimension
	int noThreads_block;				// number of threads in a block

	int N = 10;  					// size of array in each dimension
	int *a,*b,*c,*d;
	int *dev_a, *dev_b, *dev_c;
	int size;					// number of bytes in arrays

	hipEvent_t start, stop;     		// using cuda events to measure time
	float elapsed_time_ms;       		// which is applicable for asynchronous code also

/* --------------------ENTER INPUT PARAMETERS AND DATA -----------------------*/

do {  // loop to repeat complete program

	printf ("Device characteristics -- some limitations (compute capability 1.0)\n");
	printf ("		Maximum number of threads per block = 512\n");
	printf ("		Maximum sizes of x- and y- dimension of thread block = 512\n");
	printf ("		Maximum size of each dimension of grid of thread blocks = 65535\n");
	
	printf("Enter size of array in one dimension (square array), currently %d\n",N);
	scanf("%d",&N);
		
	do {
		printf("\nEnter nuumber of blocks per grid in x dimension), currently %d  : ",Grid_Dim_x);
		scanf("%d",&Grid_Dim_x);

		printf("\nEnter nuumber of blocks per grid in y dimension), currently %d  : ",Grid_Dim_y);
		scanf("%d",&Grid_Dim_y);

		printf("\nEnter nuumber of threads per block in x dimension), currently %d  : ",Block_Dim_x);
		scanf("%d",&Block_Dim_x);

		printf("\nEnter nuumber of threads per block in y dimension), currently %d  : ",Block_Dim_y);
		scanf("%d",&Block_Dim_y);

		noThreads_x = Grid_Dim_x * Block_Dim_x;		// number of threads in x dimension
		noThreads_y = Grid_Dim_y * Block_Dim_y;		// number of threads in y dimension

		noThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

		if (noThreads_x < N) printf("Error -- number of threads in x dimension less than number of elements in arrays, try again\n");
		else if (noThreads_y < N) printf("Error -- number of threads in y dimension less than number of elements in arrays, try again\n");
		else if (noThreads_block > 512) printf("Error -- too many threads in block, try again\n");
		else printf("Number of threads not used = %d\n", noThreads_x * noThreads_y - N * N);

	} while (noThreads_x < N || noThreads_y < N || noThreads_block > 512);

	dim3 Grid(Grid_Dim_x, Grid_Dim_x);		//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device

	size = N * N * sizeof(int);		// number of bytes in total in arrays

	a = (int*) malloc(size);		//this time use dynamically allocated memory for arrays on host
	b = (int*) malloc(size);
	c = (int*) malloc(size);		// results from GPU
	d = (int*) malloc(size);		// results from CPU

	for(i=0;i < N;i++)			// load arrays with some numbers
	for(j=0;j < N;j++) {
		a[i * N + j] = i;
		b[i * N + j] = i;
	}

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

	hipMalloc((void**)&dev_a, size);		// allocate memory on device
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , size ,hipMemcpyHostToDevice);

	hipEventCreate(&start);     		// instrument code to measure start time
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
//	cudaEventSynchronize(start);  	// Needed?

	gpu_matrixadd<<<Grid,Block>>>(dev_a,dev_b,dev_c,N);

	hipMemcpy(c,dev_c, size ,hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

//	for(i=0;i < N;i++) 
//	for(j=0;j < N;j++)
//	   printf("%d+%d=%d\n",a[i * N + j],b[i * N + j],c[i * N + j]);

	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);  // print out execution time

/* ------------- COMPUTATION DONE ON HOST CPU ----------------------------*/

	hipEventRecord(start, 0);		// use same timing
//	cudaEventSynchronize(start);  	// Needed?

	cpu_matrixadd(a,b,d,N);		// do calculation on host

	hipEventRecord(stop, 0);     	// instrument code to measue end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms);  // print out execution time

/* ------------------- check device creates correct results -----------------*/

	for(i=0;i < N*N;i++) {
		if (c[i] != d[i]) printf("*********** ERROR in results, CPU and GPU create different answers ********\n");
		break;
	}

	printf("\nEnter c to repeat, return to terminate\n");
	scanf("%c",&key);
	scanf("%c",&key);

} while (key == 'c'); // loop of complete program

/* --------------  clean up  ---------------------------------------*/
	free(a);
	free(b);
	free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}



