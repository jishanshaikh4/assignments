#include "hip/hip_runtime.h"
// Program for Finding Maximum element in CUDA using Reduction technique
// For Hadoop-CUDA Lab
// NOTE: THIS PROGRAM USES SOME DEPRECATED FUNCTIONS; HENCE THE WARNINGS!

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Thread block size
#define BLOCK_SIZE 512

//  Size of Array
#define SOA 8192

// Allocates an array with random integer entries.
void randomInit(int* data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = rand_r();
}

__global__ void ReductionMax2(int *input, int *results, int n)    //take thread divergence into account
{	
	extern __shared__ int sdata[]; 
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tx = threadIdx.x; 
	 //load input into __shared__ memory 
	int x = INT_MIN; 
	if(i < n) 
		x = input[i]; 
	sdata[tx] = x; 
	__syncthreads();

	// block-wide reduction
	for(unsigned int offset = blockDim.x>>1; offset > 0; offset >>= 1)
	{
		__syncthreads();
		if(tx < offset)
	    {
			if(sdata[tx + offset] > sdata[tx])
				sdata[tx] = sdata[tx + offset];
		}

	}

		// finally, thread 0 writes the result 
	if(threadIdx.x == 0) 
	{ 
		// the result is per-block 
		results[blockIdx.x] = sdata[0]; 
	} 
}


// get global max element via per-block reductions 
	int main() 
	{ 
		int num_blocks = SOA / BLOCK_SIZE;
		
		//allocate host memory for array a
		unsigned int mem_size_a = sizeof(int) * SOA;
		int* h_a = (int*)malloc(mem_size_a);
		
		//initialize host memory
		randomInit(h_a,SOA);

		//allocate device memory
		int* d_a;
		hipMalloc((void**) &d_a, mem_size_a);

		//copy host memory to device
		hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice);

		//allocate device memory for temporary results
		unsigned int mem_size_b = sizeof(int) * SOA;
		int* d_b;
		hipMalloc((void**) &d_b, mem_size_b);
		
		//allocate device memory for final result
		unsigned int mem_size_c = sizeof(int) * num_blocks;
		int* d_c;
		hipMalloc((void**) &d_c, mem_size_c);	

		//setup execution parameters
		dim3 block(1,BLOCK_SIZE);
		dim3 grid(4,4);

		//execute the kernel
		//first reduce per-block partial maxs
		ReductionMax2<<<grid, block>>>(d_a,d_b,SOA);
		
		//then reduce partial maxs to a final max
		ReductionMax2<<<grid, block>>>(d_b,d_c,num_blocks);

       	// allocate host memory for the result
		int* h_c = (int*)malloc(mem_size_c);

		//copy final result from device to host
		hipMemcpy(h_c, d_c, mem_size_c, hipMemcpyDeviceToHost);

		//clean up memory
		free(h_a);
		free(h_c);
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		hipDeviceReset();

}
